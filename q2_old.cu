#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include "assert.h"
using namespace std;

std::vector<int> readFile(string filename)
{
    ifstream infile (filename);
    vector<int> vnum;
    string line;
    int index = 0;

    while(getline(infile, line))
    {
        stringstream ss (line);
        string sint;
        while(getline(ss, sint, ','))
        {
            vnum.push_back(stoi(sint));
            index += 1;
        }
    }

    return vnum;
}

__global__ void q1a_min(int* gpu_out, int* gpu_in, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int min = INT_MAX;
    for(int i=0; i<n;i++)
    {
        if(gpu_in[i] < min)
            min = gpu_in[i];
    }
    if(idx == 0)
        gpu_out[0] = min;
}

__global__ void q1b_ones_digit(int* gpu_out, int* gpu_in, int n)
{
    // int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // if(idx >= n)
    //     return;
    // gpu_out[idx] = gpu_in[idx] % 10;

    // int start = (blockIdx.x * blockDim.x);
    // int stop = start + blockDim.x;
    // if(stop > n)
    //     stop = n;
    // for(int i=start;i<stop;i++)
    // {
    //     gpu_out[i] = gpu_in[i] % 10;
    // }
    for(int i=0; i<n;i++)
    {
        gpu_out[i] = gpu_in[i] % 10;
    }
}

__global__ void q2a_global_counter(int* gpu_out, int* gpu_in, int n)
{
    for(int i=0; i<n; i++)
    {
        int hundreds_value = gpu_in[i] / 100;
        if(hundreds_value == blockIdx.x)
        {
            gpu_out[hundreds_value] += 1;
        }
    }
}

__global__ void q2b_shared_mem_counter(int* gpu_out, int* gpu_in, int n)
{
    extern __shared__ int shared_in[];
    extern __shared__ int shared_out;
    int tidx = threadIdx.x;
    // int idx = tidx + blockIdx.x * blockDim.x;
    
    shared_out = 0;
    if(tidx == 0)
    {
        
        for(int i=0;i<n;i++)
        {
            shared_in[i] = gpu_in[i];
        }
    }
    __syncthreads(); // make sure entire block is loaded!


    for(int i=0;i<n;i++)
    {
        int hundreds_value = shared_in[i] / 100;
        if(hundreds_value == blockIdx.x)
            shared_out += 1;
    }
    __syncthreads();
    if(tidx == 0)
    {
        gpu_out[blockIdx.x] = shared_out;
    }

    __syncthreads();  
}


__global__ void q2c_prll_prfx_scan(int* gpu_out, int* gpu_in, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n)
    {
        return;
    }
    int total = 0;
    for (int i = 0; i <= idx; i++)
    {
        total += gpu_in[i];
    }
    gpu_out[idx] = total;
}


int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);

    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d: %s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               dev, devProps.name, (int)devProps.totalGlobalMem,
               (int)devProps.major, (int)devProps.minor,
               (int)devProps.clockRate);
    }

    vector<int> vnum = readFile("inp.txt");
    const int IN_SIZE = vnum.size();
    const int IN_BYTES = IN_SIZE * sizeof(int);
    const int OUT_SIZE = 10; //this is specific to the output range.
    const int OUT_BYTES = OUT_SIZE * sizeof(int);

    int* numbers;
    numbers = (int *)malloc(IN_BYTES);
    for(int i=0; i < vnum.size(); i++)
        numbers[i] = vnum[i];

    // const int maxThreadsPerBlock = 512;
    int blocks = 10;
    int threads = 8;
    // found it to be consistent at 8 (local GTX 1070Ti).
    // Found inconsistencies at 16 threads per block.
    // int threads = 512; // max threads
    printf("Input size: %d   blocks: %d   threads: %d\n\n", IN_SIZE, blocks, threads);


    int *gpu_in;
    int *gpu_out_1a;
    int *gpu_out_1b;
    int *gpu_out_2a;
    int *gpu_out_2b;
    int *gpu_out_2c;

    int cpu_out_1a[OUT_SIZE] = {0};
    // int* cpu_out_1b;
    // cpu_out_1b = (int *)malloc(IN_SIZE);
    int cpu_out_1b[10000] = {0}; // TODO!! NEED TO CHANGE!!!
    int cpu_out_2a[OUT_SIZE] = {0};
    int cpu_out_2b[OUT_SIZE] = {0};
    int cpu_out_2c[OUT_SIZE] = {0};

    hipError_t ret;

    float elapsedTime_1a;
    float elapsedTime_1b;
    float elapsedTime_2a;
    float elapsedTime_2b;
    float elapsedTime_2c;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    

    ret = hipMalloc((void **) &gpu_in, IN_BYTES);
    printf("gpu_in Malloc %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMalloc((void **) &gpu_out_1a, IN_BYTES);
    printf("gpu_out_1a Malloc %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMalloc((void **) &gpu_out_1b, IN_BYTES);
    printf("gpu_out_1b Malloc %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMalloc((void **) &gpu_out_2a, OUT_BYTES);
    printf("gpu_out_2a Malloc %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMalloc((void **) &gpu_out_2b, OUT_BYTES);
    printf("gpu_out_2b Malloc %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMalloc((void **) &gpu_out_2c, OUT_BYTES);
    printf("gpu_out_2c Malloc %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));

    ret = hipMemcpy((void *)gpu_in, (void *)numbers, IN_BYTES , hipMemcpyHostToDevice);
    printf("gpu_in Memcpy %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));


    // see https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#execution-configuration 
    // for <<<Dg, Db, Ns, S>>> parameter explanation.

    hipEventRecord(start, 0);
    q1a_min<<<blocks, threads>>>(gpu_out_1a, gpu_in, IN_SIZE);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime_1a, start, stop);


    hipEventRecord(start, 0);
    q1b_ones_digit<<<blocks, threads>>>(gpu_out_1b, gpu_in, IN_SIZE);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime_1b, start, stop);


    hipEventRecord(start, 0);
    q2a_global_counter<<<blocks, threads>>>(gpu_out_2a, gpu_in, IN_SIZE);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime_2a, start, stop);

    hipEventRecord(start, 0);
    q2b_shared_mem_counter<<<blocks, threads, (IN_SIZE+OUT_SIZE)*sizeof(int)>>>(gpu_out_2b, gpu_in, IN_SIZE);    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime_2b, start, stop);

    hipEventRecord(start, 0);
    q2c_prll_prfx_scan<<<blocks, threads>>>(gpu_out_2c, gpu_out_2a, IN_SIZE);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime_2c, start, stop);


    ret = hipMemcpy(cpu_out_1a, gpu_out_1a, OUT_BYTES, hipMemcpyDeviceToHost);
    printf("cpu_out_1a Memcpy %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMemcpy(cpu_out_1b, gpu_out_1b, IN_BYTES, hipMemcpyDeviceToHost);
    printf("cpu_out_1b Memcpy %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMemcpy(cpu_out_2a, gpu_out_2a, OUT_BYTES, hipMemcpyDeviceToHost);
    printf("cpu_out_2a Memcpy %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMemcpy(cpu_out_2b, gpu_out_2b, OUT_BYTES, hipMemcpyDeviceToHost);
    printf("cpu_out_2b Memcpy %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    ret = hipMemcpy(cpu_out_2c, gpu_out_2c, OUT_BYTES, hipMemcpyDeviceToHost);
    printf("cpu_out_2c Memcpy %s\n", ret == hipSuccess ? "Success!": hipGetErrorString(ret));
    

    // correct output:
    // 2a & b): 510, 1095, 1051, 1035, 1063, 1012, 1067, 1053, 1053, 1061
    // 2c) 510, 1605, 2656, 3691, 4754, 5766, 6833, 7886, 8939, 10000


    printf("\n\n1a:   %f\n", elapsedTime_1a);
    printf("Minimum = %d", cpu_out_1a[0]);



    int correct_count = 0;
    printf("\n\n1b:   %f\n", elapsedTime_1b);
    for(int i=0;i<IN_SIZE;i++)
    {
        // printf("%d ", cpu_out_1b[i]);
        if(numbers[i] % 10 == cpu_out_1b[i])
        {
            correct_count += 1;
        //     // printf("%d vs %d\n", numbers[i], cpu_out_1b[i]);
        }
    }
    printf("\nCount with correct ones digit:   %d / %d", correct_count, IN_SIZE);

    printf("\n\n2a:   %f\n", elapsedTime_2a);
    for(int i=0;i<OUT_SIZE;i++)
    {
        printf("%d=%d  ", i, cpu_out_2a[i]);
    }

    printf("\n\n2b:   %f\n", elapsedTime_2b);
    for(int i=0;i<OUT_SIZE;i++)
    {
        printf("%d=%d  ", i, cpu_out_2b[i]);
    }


    printf("\n\n2c:   %f\n", elapsedTime_2c);
    for(int i=0;i<OUT_SIZE;i++)
    {
        printf("%d=%d  ", i, cpu_out_2c[i]);
    }

    printf("\n\n");

    hipFree(gpu_in);
    hipFree(gpu_out_2a);
    hipFree(gpu_out_2b);
    hipFree(gpu_out_2c);
    return 0;
}

